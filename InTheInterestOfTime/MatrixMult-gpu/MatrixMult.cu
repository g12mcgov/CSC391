#include "hip/hip_runtime.h"
#define TILE_WIDTH 2

#include <stdio.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>

__global__ void matrixMult (float *A, float *B, float *C, int width);

/*
* random_populate(float *, int)
*
* Populates a matrix with random floats
*/
void random_populate(float *matrix, int size) {
   	for (int i = 0; i < size; i++) {
   		float random = rand() % 20;
   		matrix[i] = random;
   	}
}

/*
* print_matrix(int, int, float *)
*
* Prints a matrix in user friendly format
*/
void print_matrix(int rows, int columns, float *matrix) {
	for(int i = 0; i < rows; i++) {
		for(int j = 0; j < columns; j++) {
			printf("%lf\t", matrix[i * columns + j]);
		}
		printf("\n");
	}
}

/*
* check_command_line_args(int, char *)
*
* Checks to see whether used used proper
* number of command line arguments.
*/
void check_command_line_args(int argc, char *argv[]) {
	/* 
	* Ensure command line args are limited to only 3
	* (Excluding program name)
	*/
	if(argc != 5) {
		printf("Invalid Number of Arguments\n");
		exit(1);
	}
	// Check matrices are valid for multiplication
	if(atoi(argv[2]) != atoi(argv[3])) {
		printf("Columns of Matrix A do not match rows of Matrix B\n");
		exit(1);
	}
	// Check command line args are positive
	int i = 0;
	for(i = 0; i < argc; i++) {
		// We have a negative argument
		if(atoi(&argv[i][0]) < 0) {
			printf("Invalid Argument: Negative Number given for Matrix Dimensions \n");
			exit(1);
		}
	}
}

/*
* write_to_file(int, int, float *)
*
* Writes a matrix to a '.dat' file
*/
void write_to_file(int matrixC_rows, int matrixC_columns, float *C) {
	FILE *fp;
	fp = fopen("./product.dat", "w");

	// Error opening file
	if(fp == NULL) {
		printf("Error opening file for writing");
		exit(1);
	}
	fprintf(fp, "\nMatrix Product:\n");
	// Write Matrix to file
	for(int i = 0; i < matrixC_rows; i++) {
		for(int j = 0; j < matrixC_columns; j++) {
			fprintf(fp, "%lf\t", C[i * matrixC_columns + j]);
		}
		fprintf(fp, "\n");
	}
	// Close file
	fclose(fp);
}

/*
* MAIN
*/
int main(int argc, char *argv[]) {
	// Seed random time agaist sys time
	srand(time(NULL));

	// Check command line arguments
	check_command_line_args(argc, argv);
	
	// Get Matrix A dimensions
    int matrixA_rows = atoi(argv[1]);
    int matrixA_columns = atoi(argv[2]);
    
    // Get Matrix B dimensions
    int matrixB_rows = atoi(argv[3]);
    int matrixB_columns = atoi(argv[4]);

    // Product Matrix dimensions
    int matrixC_rows = matrixA_columns;
	int matrixC_columns = matrixB_rows;

    // Get size of memory needed
    int size_A = sizeof(float) * matrixA_rows * matrixA_columns;
    int size_B = sizeof(float) * matrixB_rows * matrixB_columns;

    // Declare Host Matrices
	float *A = (float*)malloc(size_A);
	float *B = (float*)malloc(size_B);

	// Initialize host memory
	random_populate(A, matrixA_rows * matrixA_columns);
	random_populate(B, matrixB_rows * matrixB_columns);

	// Print Matrices
	printf("\nMatrix A:\n");
	print_matrix(matrixA_rows, matrixA_columns, A);
	printf("\nMatrix B:\n");
	print_matrix(matrixB_rows, matrixB_columns, B);

	// Allocate Device Memory
	float* dev_A;
	float* dev_B;
	hipMalloc((void**) &dev_A, size_A);
	hipMalloc((void**) &dev_B, size_B);

	// Copy Host to Device
	hipMemcpy(dev_A, A, size_A, hipMemcpyHostToDevice);
	hipMemcpy(dev_B, B, size_B, hipMemcpyHostToDevice);

	// Compute + Allocate size for product matrix
	unsigned int size_C = sizeof(float) * matrixC_rows * matrixC_columns;
	float* C = (float*) malloc(size_C);

	// Allocate that memory on the GPU
	float* dev_C;
	hipMalloc((void**) &dev_C, size_C);

	// setup execution parameters
	dim3 dimBlock(TILE_WIDTH, TILE_WIDTH); 
	dim3 dimGrid((int)ceil(matrixC_columns/dimBlock.x), (int)ceil(matrixC_columns/dimBlock.y));

	// execute the kernel
	matrixMult<<< dimGrid, dimBlock >>>(dev_A, dev_B, dev_C, matrixC_columns);

	hipMemcpy(C, dev_C, size_C, hipMemcpyDeviceToHost);

	// Print product matrix
	printf("\nMatrix C:\n");
	print_matrix(matrixC_rows, matrixC_columns, C);

	// Write to file
	write_to_file(matrixC_rows, matrixC_columns, C);

	// Free memory on host + GPU
	free(A);
	free(B);
	free(C);
	hipFree(dev_A);
	hipFree(dev_B);
	hipFree(dev_C); 
}

/*
* matrixMult(float *, float *, float *, int)
*
* Performs matrix multiplication for matrices of floats
*/
__global__ void matrixMult(float *A, float *B, float *C, int width) {
 	float sum = 0;
 
	int col = blockIdx.x * TILE_WIDTH + threadIdx.x;
	int row = blockIdx.y * TILE_WIDTH + threadIdx.y;
	int k = 0;

	if(col < width && row < width) {
		for (k = 0; k < width; k++) {
			sum += A[row * width + k] * B[k * width + col];
		}
		C[row * width + col] = sum;
	}
}