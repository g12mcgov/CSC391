#include "hip/hip_runtime.h"
#define TILE_WIDTH 2

#include <stdio.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>

__global__ void matrixMult (float *A, float *B, float *C, int width);

// Allocates a matrix with random float entries.
void randomInit(float *matrix, int size) {
   	for (int i = 0; i < size; i++) {
   		float random = rand() % 20;
   		matrix[i] = random;
   	}
}

void print_matrix(int rows, int columns, float *matrix) {
	for(int i = 0; i < rows; i++) {
		for(int j = 0; j < columns; j++) {
			printf("%lf\t", matrix[i * columns + j]);
		}
		printf("\n");
	}
}

/*
* check_command_line_args(int)
*
* Checks to see whether used used proper
* number of command line arguments.
*/
void check_command_line_args(int argc, char *argv[]) {
	/* 
	* Ensure command line args are limited to only 3
	* (Excluding program name)
	*/
	if(argc != 5) {
		printf("Invalid Number of Arguments\n");
		exit(1);
	}
	// Check matrices are valid for multiplication
	if(atoi(argv[2]) != atoi(argv[3])) {
		printf("Columns of Matrix A do not match rows of Matrix B\n");
		exit(1);
	}
	// Check command line args are positive
	int i = 0;
	for(i = 0; i < argc; i++) {
		// We have a negative argument
		if(atoi(&argv[i][0]) < 0) {
			printf("Invalid Argument: Negative Number given for Matrix Dimensions \n");
			exit(1);
		}
	}
}

void write_to_file(int matrixC_rows, int matrixC_columns, int *C) {
	FILE *fp;
	fp = fopen("./product.dat", "w");

	// Error opening file
	if(fp == NULL) {
		printf("Error opening file for writing");
		exit(1);
	}
	fprintf(fp, "\nMatrix Product:\n");
	// Write Matrix to file
	for(int i = 0; i < matrixC_rows; i++) {
		for(int j = 0; j < matrixC_columns; j++) {
			fprintf(fp, "%d\t", C[i * matrixC_columns + j]);
		}
		fprintf(fp, "\n");
	}
	// Close file
	fclose(fp);
}

int * populate_matrix(int *matrix, int matrix_rows, int matrix_columns) {
	// Seed random against sys clock
	srand(time(NULL));
	// Indeces
	int i = 0;
	int j = 0;

	// Populate Matrix A
	for(i = 0; i < matrix_rows; i++) {
		for(j = 0; j < matrix_columns; j++) {
			matrix[i * matrix_columns + j] = rand() % 20;
		}
	}
}

int main(int argc, char *argv[]) {
	srand(time(NULL));

	// Check command line arguments
	check_command_line_args(argc, argv);
	
	// Get Matrix A dimensions
    int matrixA_rows = atoi(argv[1]);
    int matrixA_columns = atoi(argv[2]);
    
    // Get Matrix B dimensions
    int matrixB_rows = atoi(argv[3]);
    int matrixB_columns = atoi(argv[4]);

    // Product Matrix dimensions
    int matrixC_rows = matrixA_columns;
	int matrixC_columns = matrixB_rows;

    // Get size of memory needed
    int size_A = sizeof(float) * matrixA_rows * matrixA_columns;
    int size_B = sizeof(float) * matrixB_rows * matrixB_columns;

    // Declare Host Matrices
	float *A = (float*)malloc(size_A);
	float *B = (float*)malloc(size_B);

	// 2. initialize host memory
	randomInit(A, matrixA_rows * matrixA_columns);
	randomInit(B, matrixB_rows * matrixB_columns);

	print_matrix(matrixA_rows, matrixA_columns, A);
	print_matrix(matrixB_rows, matrixB_columns, B);

	// Allocate Device Memory
	float* dev_A;
	float* dev_B;
	hipMalloc((void**) &dev_A, size_A);
	hipMalloc((void**) &dev_B, size_B);

	// Copy Host to Device
	hipMemcpy(dev_A, A, size_A, hipMemcpyHostToDevice);
	hipMemcpy(dev_B, B, size_B, hipMemcpyHostToDevice);

	// 4. allocate host memory for the result C
	unsigned int size_C = sizeof(float) * matrixC_rows * matrixC_columns;
	float* C = (float*) malloc(size_C);

	float* dev_C;
	hipMalloc((void**) &dev_C, size_C);

	// 5. perform the calculation
	// setup execution parameters
	dim3 dimBlock(TILE_WIDTH, TILE_WIDTH); 
	dim3 dimGrid((int)ceil(matrixC_columns/dimBlock.x), (int)ceil(matrixC_columns/dimBlock.y));

	// execute the kernel
	matrixMult<<< dimGrid, dimBlock >>>(dev_A, dev_B, dev_C, matrixC_columns);

	// 11. copy result from device to host
	hipMemcpy(C, dev_C, size_C, hipMemcpyDeviceToHost);

	print_matrix(matrixC_rows, matrixC_columns, C);

	// 7. clean up memory
	free(A);
	free(B);
	free(C);
	hipFree(dev_A);
	hipFree(dev_B);
	hipFree(dev_C); 

 //    // Indices
 //    int i = 0;
 //    int j = 0;

 //    A = (int *)malloc(matrixA_rows * matrixA_columns * sizeof(int));
 //    B = (int *)malloc(matrixB_rows * matrixB_columns * sizeof(int));

	// A = populate_matrix(A, matrixA_rows, matrixA_columns);
	// B = populate_matrix(B, matrixB_rows, matrixB_columns);

	// // Dimensions for product matrix
	// int matrixC_rows = matrixA_columns;
	// int matrixC_columns = matrixB_rows;

	// // Initialize C matrix
	// C = (int*)calloc(matrixC_columns * matrixC_columns, sizeof(int));
	// for(i = 0; i < matrixC_rows; i++) {
	// 	for(j = 0; j < matrixC_columns; j++) {
	// 		C[i * matrixC_columns + j] = 0;
	// 	}
	// }	

	// dim3 dimBlock(TILE_WIDTH, TILE_WIDTH);
	// dim3 dimGrid((int)ceil(matrixC_rows/ dimBlock.x), (int)ceil(matrixC_columns / dimBlock.y));

	// // Declare Device Matrices
 //    int *dev_a = (int *)malloc(matrixA_rows * matrixA_columns * sizeof(int));
 //    int *dev_b = (int *)malloc(matrixB_rows * matrixB_columns * sizeof(int));
 //    int *dev_c = (int *)calloc(matrixC_rows * matrixC_columns, sizeof(int));

	// int size = matrixC_columns * matrixC_rows * sizeof(int);

	// // initialize a and 8 ma8rices here
	// hipMalloc(&dev_a, size);
	// hipMalloc(&dev_b, size);
	// hipMalloc(&dev_c, size);

	// hipMemcpy(dev_a, A, size, hipMemcpyHostToDevice);
	// hipMemcpy(dev_b, B, size, hipMemcpyHostToDevice);
	// hipMemcpy(dev_c, C, size, hipMemcpyHostToDevice);
}

	//matrixMult<<<dimGrid, dimBlock>>>(dev_a, dev_b, dev_c, matrixC_columns);

	// hipDeviceSynchronize();

	// hipMemcpy(C, dev_c, size, hipMemcpyDeviceToHost);

	// //printf(c);
	// for(i = 0; i < matrixC_rows; i++) {
	// 	for(j = 0; j < matrixC_columns; j++) {
	// 		printf("%d", C[i][j]);
	// 		printf("\t");
	// 	}
	// 	printf("\n");
	// }

	// hipFree(dev_a);
	// hipFree(dev_b);
	// hipFree(dev_c);

__global__ void matrixMult(float *A, float *B, float *C, int width) {
 	float sum = 0;
 
	int col = blockIdx.x * TILE_WIDTH + threadIdx.x;
	int row = blockIdx.y * TILE_WIDTH + threadIdx.y;
	int k = 0;

	if(col < width && row < width) {
		for (k = 0; k < width; k++) {
			sum += A[row * width + k] * B[k * width + col];
		}
		C[row * width + col] = sum;
	}
}